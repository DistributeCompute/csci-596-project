#include "hip/hip_runtime.h"
include <stdio.h>
#include <mpi.h>
#include <omp.h>
#include <hip/hip_runtime.h>
#include <math.h>

#define N 100000000  // Number of terms in the harmonic series
#define NUM_DEVICE 2 // Number of GPU devices = Number of OpenMP threads

// CUDA kernel to compute partial harmonic series
__global__ void harmonic_sum(double *sum, int thread_id_global, int nthreads_total, int nterms) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;  // Global thread index
    if (idx < nterms) {
        int k = thread_id_global + 1 + idx * nthreads_total;  // Interleaved term index
        if (k <= N) {
            sum[idx] = 1.0 / k;  // Compute 1/k
        } else {
            sum[idx] = 0.0;  // Handle out-of-bound terms
        }
    }
}

int main(int argc, char **argv) {
    int myid, nproc;
    double harmonic_sum_local = 0.0, harmonic_sum_global, euler_mascheroni_constant;
    double *sumHost, *sumDev;  // Pointers to host & device arrays
    int dev_used;

    MPI_Init(&argc, &argv);
    MPI_Comm_rank(MPI_COMM_WORLD, &myid);  // My MPI rank
    MPI_Comm_size(MPI_COMM_WORLD, &nproc);  // Number of MPI processes

    omp_set_num_threads(NUM_DEVICE); // One OpenMP thread per GPU device

    int nthreads_total = nproc * NUM_DEVICE; // Total number of compute units

    #pragma omp parallel private(sumHost, sumDev, dev_used) reduction(+:harmonic_sum_local)
    {
        int mpid = omp_get_thread_num(); // OpenMP thread ID
        int thread_id_global = myid * NUM_DEVICE + mpid; // Global thread ID

        // Set the CUDA device for the thread
        hipSetDevice(mpid % NUM_DEVICE);

        // Calculate the number of terms this thread will process
        int nterms = (N - thread_id_global + nthreads_total - 1) / nthreads_total;

        size_t size = nterms * sizeof(double);  // Memory size for this thread
        sumHost = (double *)malloc(size);       // Allocate host memory
        hipMalloc((void **)&sumDev, size);     // Allocate device memory

        // Adjust the number of threads and blocks based on nterms
        int threadsPerBlock = 256;
        int blocksPerGrid = (nterms + threadsPerBlock - 1) / threadsPerBlock;

        // Launch the CUDA kernel
        harmonic_sum <<<blocksPerGrid, threadsPerBlock>>> (sumDev, thread_id_global, nthreads_total, nterms);

        // Copy the results back to host
        hipMemcpy(sumHost, sumDev, size, hipMemcpyDeviceToHost);

        // Sum up the partial results
        double partial_sum = 0.0;
        for (int i = 0; i < nterms; i++)
            partial_sum += sumHost[i];
	        // Accumulate the partial sums into the local total
        harmonic_sum_local += partial_sum;

        // CUDA cleanup
        free(sumHost);
        hipFree(sumDev);
        hipGetDevice(&dev_used);

        // Output the partial sum for this thread
        printf("myid = %d; mpid = %d: device used = %d; partial sum = %.20f\n", myid, mpid, dev_used, partial_sum);
    } // End of OpenMP parallel region

    // Reduction over MPI processes
    MPI_Allreduce(&harmonic_sum_local, &harmonic_sum_global, 1, MPI_DOUBLE, MPI_SUM, MPI_COMM_WORLD);

    // Compute the Euler-Mascheroni constant
    if (myid == 0) {
        double lnN = log((double)N);
        euler_mascheroni_constant = harmonic_sum_global - lnN;
        printf("Euler-Mascheroni Constant (γ) ≈ %.20f\n", euler_mascheroni_constant);
    }

    MPI_Finalize();
    return 0;
}


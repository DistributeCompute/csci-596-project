#include "hip/hip_runtime.h"
// DistributeCompute: Parallel Computation using MPI and CUDA
// Apery's Constant

// Hybrid MPI+CUDA computation of Pi
#include <stdio.h>
#include <mpi.h>
#include <omp.h>
#include <hip/hip_runtime.h>
#include <math.h>

#define NUM_DEVICE 2 // # of GPU devices = # of OpenMP threads
#define NBIN  100000000  // Number of bins, increasing for accuracy

__global__ void cal_apery(double *sum, int thread_id_global, int nthreads_total, int nterms) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;  // Global index 
    if (idx < nterms) {
        // Compute the actual term index n for this thread
        int n = thread_id_global + 1 + idx * nthreads_total;
        if (n <= NBIN) {
            double dn = (double)n;
            sum[idx] = 1.0/(dn*dn*dn);
        } else {
            sum[idx] = 0.0;
        }
    }
}


int main(int argc, char **argv) {
    int myid, nproc;
    double apery = 0.0, apery_final;
    double *sumHost, *sumDev;

    MPI_Init(&argc, &argv);
    MPI_Comm_rank(MPI_COMM_WORLD, &myid);  // MPI rank
    MPI_Comm_size(MPI_COMM_WORLD, &nproc); // Number of MPI processes

    // Set number of threads = number of GPUs per MPI rank
    omp_set_num_threads(NUM_DEVICE);

    int nthreads_total = nproc * NUM_DEVICE; // Total number of (MPI×OMP) "threads"

    #pragma omp parallel private(sumHost, sumDev) reduction(+:apery)
    {
        int mpid = omp_get_thread_num(); // OMP thread id
        int thread_id_global = myid * NUM_DEVICE + mpid; // Global thread ID across all ranks and threads

        // Assign GPU device
        hipSetDevice(mpid % NUM_DEVICE);

        // Number of terms this particular thread will handle
        int nterms = (NBIN - thread_id_global + nthreads_total - 1) / nthreads_total;
        if (nterms < 0) nterms = 0; // Safety check

        size_t size = nterms * sizeof(double);
        sumHost = (double *)malloc(size);
        hipMalloc((void **)&sumDev, size);

        int threadsPerBlock = 256;
        int blocksPerGrid = (nterms + threadsPerBlock - 1) / threadsPerBlock;

        // Launch CUDA kernel
        cal_apery<<<blocksPerGrid, threadsPerBlock>>>(sumDev, thread_id_global, nthreads_total, nterms);

        // Copy results back to host
        hipMemcpy(sumHost, sumDev, size, hipMemcpyDeviceToHost);

        // Sum partial results on host
        double partial_sum = 0.0;
        for (int i = 0; i < nterms; i++)
            partial_sum += sumHost[i];

        apery += partial_sum;

        int dev_used;
        hipGetDevice(&dev_used);

        // Debug print
        printf("myid = %d; mpid = %d; device = %d; partial zeta(3) sum = %.15f\n", myid, mpid, dev_used, partial_sum);

        // Cleanup
        free(sumHost);
        hipFree(sumDev);
    }

    // MPI reduce to get global sum
    MPI_Allreduce(&apery, &apery_final, 1, MPI_DOUBLE, MPI_SUM, MPI_COMM_WORLD);

    if (myid == 0) {
        printf("Apery’s constant apery(3) ≈ %.30f\n", apery_final);
    }

    MPI_Finalize();
    return 0;
}

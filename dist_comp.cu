#include "hip/hip_runtime.h"
// DistributeCompute: Parallel Computation using MPI and CUDA

// Hybrid MPI+CUDA computation of Pi
#include <stdio.h>
#include <mpi.h>
#include <omp.h>
#include <hip/hip_runtime.h>
#include <math.h>

#define NUM_DEVICE 2 // # of GPU devices = # of OpenMP threads
#define NBIN  100000000  // Number of bins, increasing for accuracy

__global__ void cal_apery(double *sum, int thread_id_global, int nthreads_total, int nterms) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;  // Global index 
    if (idx < nterms) {
        // Compute the actual term index n for this thread
        int n = thread_id_global + 1 + idx * nthreads_total;
        if (n <= NBIN) {
            double dn = (double)n;
            sum[idx] = 1.0/(dn*dn*dn);
        } else {
            sum[idx] = 0.0;
        }
    }
}



// int main(int argc,char **argv) {
// 	int myid,nproc,nbin,tid;
// 	float step,offset,pi=0.0,pig;
	
// 	float apery=0.0, apery_final;

// 	dim3 dimGrid(NUM_BLOCK,1,1);  // Grid dimensions (only use 1D)
// 	dim3 dimBlock(NUM_THREAD,1,1);  // Block dimensions (only use 1D)
// 	float *sumHost,*sumDev;  // Pointers to host & device arrays
// 	int dev_used;

// 	MPI_Init(&argc,&argv);
// 	MPI_Comm_rank(MPI_COMM_WORLD,&myid);  // My MPI rank
// 	MPI_Comm_size(MPI_COMM_WORLD,&nproc);  // Number of MPI processes
// 	// nbin = NBIN/nproc;  // Number of bins per MPI process
// 	// step = 1.0/(float)(nbin*nproc);  // Step size with redefined number of bins
// 	// offset = myid*step*nbin;  // Quadrature-point offset

// 	omp_set_num_threads(NUM_DEVICE); // One OpenMP thread per GPU device
// 	nbin = NBIN/(nproc*NUM_DEVICE); // # of bins per OpenMP thread
// 	step = 1.0/(float)(nbin*nproc*NUM_DEVICE);

// 	#pragma omp parallel private(offset, sumHost, sumDev, tid, dev_used) reduction(+:pi)
// 	{	
// 		int mpid = omp_get_thread_num();
// 		offset = (NUM_DEVICE*myid+mpid)*step*nbin; // Quadrature-point offset
// 		hipSetDevice(mpid%2);

// 		// hipSetDevice(myid%2);
// 		size_t size = NUM_BLOCK*NUM_THREAD*sizeof(float);  //Array memory size
// 		sumHost = (float *)malloc(size);  //  Allocate array on host
// 		hipMalloc((void **) &sumDev,size);  // Allocate array on device
// 		hipMemset(sumDev,0,size);  // Reset array in device to 0
// 		// Calculate on device (call CUDA kernel)
// 		cal_pi <<<dimGrid,dimBlock>>> (sumDev,nbin,step,offset,NUM_THREAD,NUM_BLOCK);
// 		// Retrieve result from device and store it in host array
// 		hipMemcpy(sumHost,sumDev,size,hipMemcpyDeviceToHost);
// 		// Reduction over CUDA threads
// 		for(tid=0; tid<NUM_THREAD*NUM_BLOCK; tid++)
// 			pi += sumHost[tid];
// 		pi *= step;
// 		// CUDA cleanup
// 		free(sumHost);
// 		hipFree(sumDev);
// 		hipGetDevice(&dev_used);
// 		printf("myid = %d; mpid = %d: device used = %d; partial pi = %f\n", myid, mpid, dev_used, pi);
// 	}
	
// 	// Reduction over MPI processes
// 	MPI_Allreduce(&pi,&pig,1,MPI_FLOAT,MPI_SUM,MPI_COMM_WORLD);
// 	if (myid==0) printf("Apery's con = %f\n",pig);

// 	MPI_Finalize();
// 	return 0;
// }

int main(int argc, char **argv) {
    int myid, nproc;
    double apery = 0.0, apery_final;
    double *sumHost, *sumDev;

    MPI_Init(&argc, &argv);
    MPI_Comm_rank(MPI_COMM_WORLD, &myid);  // MPI rank
    MPI_Comm_size(MPI_COMM_WORLD, &nproc); // Number of MPI processes

    // Set number of threads = number of GPUs per MPI rank
    omp_set_num_threads(NUM_DEVICE);

    int nthreads_total = nproc * NUM_DEVICE; // Total number of (MPI×OMP) "threads"

    #pragma omp parallel private(sumHost, sumDev) reduction(+:apery)
    {
        int mpid = omp_get_thread_num(); // OMP thread id
        int thread_id_global = myid * NUM_DEVICE + mpid; // Global thread ID across all ranks and threads

        // Assign GPU device
        hipSetDevice(mpid % NUM_DEVICE);

        // Number of terms this particular thread will handle
        int nterms = (NBIN - thread_id_global + nthreads_total - 1) / nthreads_total;
        if (nterms < 0) nterms = 0; // Safety check

        size_t size = nterms * sizeof(double);
        sumHost = (double *)malloc(size);
        hipMalloc((void **)&sumDev, size);

        int threadsPerBlock = 256;
        int blocksPerGrid = (nterms + threadsPerBlock - 1) / threadsPerBlock;

        // Launch CUDA kernel
        cal_apery<<<blocksPerGrid, threadsPerBlock>>>(sumDev, thread_id_global, nthreads_total, nterms);

        // Copy results back to host
        hipMemcpy(sumHost, sumDev, size, hipMemcpyDeviceToHost);

        // Sum partial results on host
        double partial_sum = 0.0;
        for (int i = 0; i < nterms; i++)
            partial_sum += sumHost[i];

        apery += partial_sum;

        int dev_used;
        hipGetDevice(&dev_used);

        // Debug print
        printf("myid = %d; mpid = %d; device = %d; partial zeta(3) sum = %.15f\n", myid, mpid, dev_used, partial_sum);

        // Cleanup
        free(sumHost);
        hipFree(sumDev);
    }

    // MPI reduce to get global sum
    MPI_Allreduce(&apery, &apery_final, 1, MPI_DOUBLE, MPI_SUM, MPI_COMM_WORLD);

    if (myid == 0) {
        printf("Apery’s constant apery(3) ≈ %.30f\n", apery_final);
    }

    MPI_Finalize();
    return 0;
}
